#include "hip/hip_runtime.h"
#include "cross_section.h"

void processCSData(CSData* cross_sections, string path_to_csdata) {
    ifstream ReadIn(path_to_csdata);

    // Check if file is opened successfully
    if (!ReadIn) {
        cerr << "Failed to open file: " << path_to_csdata << endl;
        return;
    }

    for (int i = 0; i < N_STEPS; i++) {
        if (ReadIn >> cross_sections[i].split_chance >> cross_sections[i].remove_chance) {
            continue;
        } else {
            cerr << "Failed to read data for step " << i << endl;
            break;
        }
    }
    // for (int i = 0; i < N_STEPS; i++) {
    //     ReadIn >> cross_sections[i].split_chance;
    //     ReadIn >> cross_sections[i].remove_chance;
    //     cout<< cross_sections[i].energy<<endl;
    //     cout<< cross_sections[i].split_chance<<endl;
    //     cout<< cross_sections[i].remove_chance<<endl;
    //     // printf("CS: %d \n", cross_sections[0].split_chance);
    // }
}

__device__ int energyToIndex(double energy){
    int energy_index = trunc(log10(energy));
    return (energy_index < 0) ? 0 : ((energy_index > 10) ? 10 : energy_index);
}
