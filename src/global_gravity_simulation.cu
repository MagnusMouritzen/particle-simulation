#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "utility.h"
#include "global_gravity_simulation.h"


struct Electron {
    public:
        float3 position;
        float weight;
        float3 velocity;
};

__global__ static void updateBranch(Electron* electrons, float deltaTime) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    electrons[i].velocity.y -= 9.82 * deltaTime;
    electrons[i].position.y += electrons[i].velocity.y * deltaTime;
    if (electrons[i].position.y <= 0){
        electrons[i].position.y = -electrons[i].position.y;
        electrons[i].velocity.y = -electrons[i].velocity.y;
    }
}

__global__ static void updateMath(Electron* electrons, float deltaTime) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    electrons[i].velocity.y -= 9.82 * deltaTime;
    electrons[i].position.y += electrons[i].velocity.y * deltaTime;
    int posSign = (electrons[0].position.y > 0.0) * 2 - 1;  // Does this branch?
    electrons[i].position.y *= posSign;
    electrons[i].velocity.y *= posSign;
}

void globalGravityRun(int N, int t, bool branch, bool verbose) {

    Electron* electrons_host = (Electron *)malloc(N * sizeof(Electron));
    for(int i=0; i<N; i++) {
        electrons_host[i].position = make_float3(randomFloat(0,5), randomFloat(10,50), 1.0);
        electrons_host[i].weight = 1.0;
    }

    Electron* electrons;
    hipMalloc(&electrons, N * sizeof(Electron));

    hipMemcpy(electrons, electrons_host, N * sizeof(Electron), hipMemcpyHostToDevice);

    int block_size = 256;
    int num_blocks = (N + block_size - 1) / block_size;

    if (verbose) printf("Time %d, position %.6f, velocity %.6f\n", 0, electrons_host[0].position.y, electrons_host[0].velocity.y);

    for (int i = 1; i < 101; i++){
        if (branch){
            updateBranch<<<num_blocks, block_size>>>(electrons, 0.1);
        }
        else {
            updateMath<<<num_blocks, block_size>>>(electrons, 0.1);
        }

        if (verbose && i % 5 == 0){
            hipMemcpy(electrons_host, electrons, N * sizeof(Electron), hipMemcpyDeviceToHost);
            printf("Time %d, position %.6f, velocity %.6f\n", i, electrons_host[0].position.y, electrons_host[0].velocity.y);
        }
    }
    hipMemcpy(electrons_host, electrons, N * sizeof(Electron), hipMemcpyDeviceToHost);
    if (verbose) printf("Time %d, position %.6f, velocity %.6f\n", t-1, electrons_host[0].position.y, electrons_host[0].velocity.y);

    

    
}
