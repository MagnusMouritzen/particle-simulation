#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "utility.h"
#include "multiply_simulation.h"

__device__ static void simulate(Electron* electrons, float deltaTime, int* n, int capacity, int i, int t){
    electrons[i].velocity.y -= 9.82 * deltaTime;
    electrons[i].position.y += electrons[i].velocity.y * deltaTime;

    if (electrons[i].position.y <= 0){
        electrons[i].position.y = -electrons[i].position.y;
        electrons[i].velocity.y = -electrons[i].velocity.y;
        if (electrons[i].velocity.x == 0){
            electrons[i].velocity.x = 1;
        }

        int new_i = atomicAdd(n, 1);
        if (new_i < capacity){
            // printf("Particle %d spawns particle %d\n", i, new_i);
            electrons[new_i].position.y = electrons[i].position.y;
            electrons[new_i].velocity.y = electrons[i].velocity.y;
            electrons[new_i].velocity.x = -electrons[i].velocity.x;
            electrons[new_i].position.x = electrons[i].position.x + electrons[new_i].velocity.x * deltaTime;
            electrons[new_i].timestamp = t;
        }
    }
    electrons[i].position.x += electrons[i].velocity.x * deltaTime;
}

__global__ static void updateNormal(Electron* electrons, float deltaTime, int* n, int start_n, int capacity) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // The thread index has passed the number of electrons. Thread returns if all electron are being handled
    if (i >= start_n) return;

    simulate(electrons, deltaTime, n, capacity, i, 0);
}

__global__ static void updateHuge(Electron* electrons, float deltaTime, int* n, int capacity, int t) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // The thread index has passed the number of electrons. Thread returns if all electron are being handled
    if (i >= *n || electrons[i].timestamp == t || electrons[i].timestamp == 0) return;

    simulate(electrons, deltaTime, n, capacity, i, t);
}

__global__ static void updateStatic(Electron* electrons, float deltaTime, int* n, int capacity, int t) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int num_blocks = gridDim.x;
    int block_size = blockDim.x;

    for (int i = thread_id; i < *n; i += num_blocks * block_size) {
        // The thread index has passed the number of electrons. Thread returns if all electron are being handled
        if (electrons[i].timestamp == t || electrons[i].timestamp == 0) return;

        simulate(electrons, deltaTime, n, capacity, i, t);
    }
}



static void log(bool verbose, int t, Electron* electrons_host, Electron* electrons, int* n_host, int* n, int capacity){
    if (!verbose || t % 10 != 0) return;
    hipMemcpy(n_host, n, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(electrons_host, electrons, *n_host * sizeof(Electron), hipMemcpyDeviceToHost);
    printf("Time %d, amount %d\n", t, *n_host);
    for(int i = 0; i < min(*n_host, capacity); i++){
        printf("%d: (%.6f, %.6f) (%.6f, %.6f)\n", i, electrons_host[i].position.x, electrons_host[i].position.y, electrons_host[i].velocity.x, electrons_host[i].velocity.y);
    }
    image(min(*n_host, capacity), electrons_host, t); // visualize a snapshot of the current positions of the particles     
    printf("\n");
}

void multiplyRun(int init_n, int capacity, int max_t, int mode, bool verbose) {
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    Electron* electrons_host = (Electron *)calloc(capacity, sizeof(Electron));
    for(int i=0; i<init_n; i++) {
        electrons_host[i].position = make_float3(250, 250, 1.0);
        electrons_host[i].weight = 1.0;
        electrons_host[i].timestamp = -1;
    }

    Electron* electrons;
    hipMalloc(&electrons, capacity * sizeof(Electron));

    hipMemcpy(electrons, electrons_host, init_n * sizeof(Electron), hipMemcpyHostToDevice);


    int block_size = 256;

    int* n_host = (int*)malloc(sizeof(int));
    int* n;
    hipMalloc(&n, sizeof(int));
    *n_host = init_n;
    hipMemcpy(n, n_host, sizeof(int), hipMemcpyHostToDevice);

    if (verbose) printf("Time %d, amount %d\n", 0, *n_host);

    switch(mode){
        case 0: { // Normal
            printf("Multiply normal\n");
            hipEventRecord(start);
            for (int t = 1; t < max_t; t++){
                int num_blocks = (*n_host + block_size - 1) / block_size;
                updateNormal<<<num_blocks, block_size>>>(electrons, 0.1, n, *n_host, capacity);
                hipMemcpy(n_host, n, sizeof(int), hipMemcpyDeviceToHost);

                log(verbose, t, electrons_host, electrons, n_host, n, capacity);
            }
            hipEventRecord(stop);
            break;
        }
        case 1: { // Huge
            printf("Multiply huge\n");
            int num_blocks = (capacity + block_size - 1) / block_size;
            hipEventRecord(start);
            for (int t = 1; t < max_t; t++) {
                updateHuge<<<num_blocks, block_size>>>(electrons, 0.1, n, capacity, t);
                log(verbose, t, electrons_host, electrons, n_host, n, capacity);
            }
            hipEventRecord(stop);
            break;
        }
        case 2: { // Static
            printf("Multiply static\n");
            int num_blocks;            
            hipDeviceGetAttribute(&num_blocks, hipDeviceAttributeMultiprocessorCount, 0);
            printf("Number of blocks: %d \n",num_blocks);
            hipEventRecord(start);
            for (int t = 1; t < max_t; t++) {
                updateStatic<<<num_blocks, block_size>>>(electrons, 0.1, n, capacity, t);
                log(verbose, t, electrons_host, electrons, n_host, n, capacity);
            }
            hipEventRecord(stop);
            break;
        }
        case 3: { // Dynamic
            printf("Multiply dynamic not implemented\n");
            break;
        }
        default:
            break;
    }

    hipMemcpy(n_host, n, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(electrons_host, electrons, *n_host * sizeof(Electron), hipMemcpyDeviceToHost);   
    hipEventSynchronize(stop); //skal det være her?

    float runtime_ms = 0;
    hipEventElapsedTime(&runtime_ms, start, stop);
    printf("Final amount of particles: %d\n", min(*n_host, capacity));
    printf("GPU time of program: %f ms\n", runtime_ms);
}
