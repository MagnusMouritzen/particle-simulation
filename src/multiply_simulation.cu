#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "utility.h"
#include "multiply_simulation.h"



__global__ static void updateNormal(Electron* electrons, float deltaTime, int* n, int start_n, int capacity) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // The thread index has passed the number of electrons. Thread returns if all electron are being handled
    if (i >= start_n) return;

    electrons[i].velocity.y -= 9.82 * deltaTime;
    electrons[i].position.y += electrons[i].velocity.y * deltaTime;

    if (electrons[i].position.y <= 0){
        electrons[i].position.y = -electrons[i].position.y;
        electrons[i].velocity.y = -electrons[i].velocity.y;
        if (electrons[i].velocity.x == 0){
            electrons[i].velocity.x = 1;
        }

        int new_i = atomicAdd(n, 1);
        if (new_i < capacity){
            printf("Particle %d spawns particle %d\n", i, new_i);
            electrons[new_i].position.y = electrons[i].position.y;
            electrons[new_i].velocity.y = electrons[i].velocity.y;
            electrons[new_i].velocity.x = -electrons[i].velocity.x;
            electrons[new_i].position.x = electrons[i].position.x + electrons[new_i].velocity.x * deltaTime;
        }
    }
    electrons[i].position.x += electrons[i].velocity.x * deltaTime;
}

__global__ static void updateHuge(Electron* electrons, float deltaTime, int* n, int capacity, int t) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // The thread index has passed the number of electrons. Thread returns if all electron are being handled
    if (i >= *n || electrons[i].timestamp == t || electrons[i].timestamp == 0) return;

    electrons[i].velocity.y -= 9.82 * deltaTime;
    electrons[i].position.y += electrons[i].velocity.y * deltaTime;

    if (electrons[i].position.y <= 0){
        electrons[i].position.y = -electrons[i].position.y;
        electrons[i].velocity.y = -electrons[i].velocity.y;
        if (electrons[i].velocity.x == 0){
            electrons[i].velocity.x = 1;
        }

        int new_i = atomicAdd(n, 1);
        if (new_i < capacity){
            printf("Particle %d spawns particle %d\n", i, new_i);
            electrons[new_i].position.y = electrons[i].position.y;
            electrons[new_i].velocity.y = electrons[i].velocity.y;
            electrons[new_i].velocity.x = -electrons[i].velocity.x;
            electrons[new_i].position.x = electrons[i].position.x + electrons[new_i].velocity.x * deltaTime;
            electrons[new_i].timestamp = t;
        }
    }
    electrons[i].position.x += electrons[i].velocity.x * deltaTime;
}

__global__ static void updateStatic(Electron* electrons, float deltaTime, int* n, int capacity, int t) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int num_blocks = gridDim.x;
    int block_size = blockDim.x;

    for (int i = thread_id; i < *n; i += num_blocks * block_size) {
        // The thread index has passed the number of electrons. Thread returns if all electron are being handled
        if (electrons[i].timestamp == t || electrons[i].timestamp == 0) return;

        electrons[i].velocity.y -= 9.82 * deltaTime;
        electrons[i].position.y += electrons[i].velocity.y * deltaTime;

        if (electrons[i].position.y <= 0){
            electrons[i].position.y = -electrons[i].position.y;
            electrons[i].velocity.y = -electrons[i].velocity.y;
            if (electrons[i].velocity.x == 0){
                electrons[i].velocity.x = 1;
            }

            int new_i = atomicAdd(n, 1);
            if (new_i < capacity){
                printf("Particle %d spawns particle %d\n", i, new_i);
                electrons[new_i].position.y = electrons[i].position.y;
                electrons[new_i].velocity.y = electrons[i].velocity.y;
                electrons[new_i].velocity.x = -electrons[i].velocity.x;
                electrons[new_i].position.x = electrons[i].position.x + electrons[new_i].velocity.x * deltaTime;
                electrons[new_i].timestamp = t;
            }
        }
        electrons[i].position.x += electrons[i].velocity.x * deltaTime;
    }

}

__global__ static void updateDynamic(Electron* electrons, float deltaTime) {
}

void multiplyRun(int init_n, int capacity, int max_t, int mode, bool verbose) {
    
    Electron* electrons_host = (Electron *)calloc(capacity, sizeof(Electron));
    for(int i=0; i<init_n; i++) {
        electrons_host[i].position = make_float3(25, 5, 1.0);
        electrons_host[i].weight = 1.0;
        electrons_host[i].timestamp = -1;
    }

    Electron* electrons;
    hipMalloc(&electrons, capacity * sizeof(Electron));

    hipMemcpy(electrons, electrons_host, init_n * sizeof(Electron), hipMemcpyHostToDevice);


    int block_size = 256;

    int* n_host = (int*)malloc(sizeof(int));
    int* n;
    hipMalloc(&n, sizeof(int));
    *n_host = init_n;
    hipMemcpy(n, n_host, sizeof(int), hipMemcpyHostToDevice);

    if (verbose) printf("Time %d, amount %d\n", 0, *n_host);

    switch(mode){
        case 0: { // Normal
            for (int t = 1; t < max_t; t++){
                int num_blocks = (*n_host + block_size - 1) / block_size;
                updateNormal<<<num_blocks, block_size>>>(electrons, 0.1, n, *n_host, capacity);
                
                hipMemcpy(n_host, n, sizeof(int), hipMemcpyDeviceToHost);

                if (verbose && t % 1 == 0){
                    hipMemcpy(electrons_host, electrons, *n_host * sizeof(Electron), hipMemcpyDeviceToHost);
                    printf("Time %d, amount %d\n", t, *n_host);
                    for(int i = 0; i < min(*n_host, capacity); i++){
                        if (i >= capacity) break;
                        printf("%d: (%.6f, %.6f) (%.6f, %.6f)\n", i, electrons_host[i].position.x, electrons_host[i].position.y, electrons_host[i].velocity.x, electrons_host[i].velocity.y);
                    }
                    image(min(*n_host, capacity), electrons_host, t); // visualize a snapshot of the current positions of the particles     
                    printf("\n");
                }
                if (*n_host >= capacity) break;
            }
            break;
        }
        case 1: { // Huge
            int num_blocks = (capacity + block_size - 1) / block_size;
            for (int t = 1; t < max_t; t++) {

                updateHuge<<<num_blocks, block_size>>>(electrons, 0.1, n, capacity, t);

                if (verbose && t % 1 == 0){
                    hipMemcpy(electrons_host, electrons, capacity * sizeof(Electron), hipMemcpyDeviceToHost);

                    int count = 0;

                    printf("Time %d, amount %d\n", t, *n_host);
                    for(int i = 0; i < capacity; i++) {
                        if (electrons_host[i].timestamp == 0) break;

                        printf("%d: (%.6f, %.6f) (%.6f, %.6f)\n", i, electrons_host[i].position.x, electrons_host[i].position.y, electrons_host[i].velocity.x, electrons_host[i].velocity.y);
                    }
                    image(min(count, capacity), electrons_host, t); // visualize a snapshot of the current positions of the particles     
                    printf("\n");
                }
            }
            break;
        }
        case 2: { // Static
            int num_blocks;            
            hipDeviceGetAttribute(&num_blocks, hipDeviceAttributeMultiprocessorCount, 0);
            printf("Number of blocks: %d \n",num_blocks);
            for (int t = 1; t < max_t; t++) {

                updateStatic<<<num_blocks, block_size>>>(electrons, 0.1, n, capacity, t);

                if (verbose && t % 10 == 0){
                    hipMemcpy(electrons_host, electrons, capacity * sizeof(Electron), hipMemcpyDeviceToHost);

                    int count = 0;

                    for(int i = 0; i < capacity; i++) {
                        if (electrons_host[i].timestamp == 0) break;
                        count++;

                        // printf("%d: (%.6f, %.6f) (%.6f, %.6f)\n", i, electrons_host[i].position.x, electrons_host[i].position.y, electrons_host[i].velocity.x, electrons_host[i].velocity.y);
                    }
                    printf("Time %d, amount %d\n", t, count);

                    // image(min(count, capacity), electrons_host, t); // visualize a snapshot of the current positions of the particles     
                    printf("\n");
                }
            }
            break;
        }
        case 3: { // Dynamic
            break;
        }
        default:
            break;
    }

    hipMemcpy(electrons_host, electrons, *n_host * sizeof(Electron), hipMemcpyDeviceToHost);   
}
