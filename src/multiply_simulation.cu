#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "utility.h"
#include "multiply_simulation.h"

__device__ static void simulate(Electron* electrons, float deltaTime, int* n, int capacity, int i, int t){
    electrons[i].velocity.y -= 9.82 * deltaTime * electrons[i].weight;
    electrons[i].position.y += electrons[i].velocity.y * deltaTime;

    if (electrons[i].position.y <= 0){
        electrons[i].position.y = -electrons[i].position.y;
        electrons[i].velocity.y = -electrons[i].velocity.y;

        int new_i = atomicAdd(n, 1);
        if (new_i < capacity){
            if (electrons[i].velocity.x >= 0){
                electrons[i].velocity.x += 10;
            }
            else{
                electrons[i].velocity.x -= 10;
            }

            //printf("Particle %d spawns particle %d\n", i, new_i);
            electrons[new_i].position.y = electrons[i].position.y;
            electrons[new_i].velocity.y = electrons[i].velocity.y;
            if (electrons[i].velocity.x >= 0){
                electrons[new_i].velocity.x = electrons[i].velocity.x - 20;
            }
            else{
                electrons[new_i].velocity.x = electrons[i].velocity.x + 20;
            }
            electrons[new_i].position.x = electrons[i].position.x + electrons[new_i].velocity.x * deltaTime;
            electrons[new_i].timestamp = t;
            electrons[new_i].weight = electrons[i].weight;
        }
    }
    else if (electrons[i].position.y >= 500){
        electrons[i].position.y = 500 - (electrons[i].position.y - 500);
        electrons[i].velocity.y = -electrons[i].velocity.y;
    }

    electrons[i].position.x += electrons[i].velocity.x * deltaTime;

    if (electrons[i].position.x <= 0){
        electrons[i].position.x = -electrons[i].position.x;
        electrons[i].velocity.x = -electrons[i].velocity.x;
        electrons[i].weight *= -1;
    }
    else if (electrons[i].position.x >= 500){
        electrons[i].position.x = 500 - (electrons[i].position.x - 500);
        electrons[i].velocity.x = -electrons[i].velocity.x;
        electrons[i].weight *= -1;
    }
}

__global__ static void updateNormal(Electron* electrons, float deltaTime, int* n, int start_n, int capacity) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // The thread index has passed the number of electrons. Thread returns if all electron are being handled
    if (i >= start_n) return;

    simulate(electrons, deltaTime, n, capacity, i, 0);
}

__global__ static void updateHuge(Electron* electrons, float deltaTime, int* n, int capacity, int t) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // The thread index has passed the number of electrons. Thread returns if all electron are being handled
    if (i >= *n || electrons[i].timestamp == t || electrons[i].timestamp == 0) return;

    simulate(electrons, deltaTime, n, capacity, i, t);
}

__global__ static void updateStatic(Electron* electrons, float deltaTime, int* n, int capacity, int t) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int num_blocks = gridDim.x;
    int block_size = blockDim.x;

    for (int i = thread_id; i < *n; i += num_blocks * block_size) {
        // The thread index has passed the number of electrons. Thread returns if all electron are being handled
        if (electrons[i].timestamp == t || electrons[i].timestamp == 0) return;

        simulate(electrons, deltaTime, n, capacity, i, t);
    }
}



static void log(int verbose, int t, Electron* electrons_host, Electron* electrons, int* n_host, int* n, int capacity){
    if (verbose == 0 || t % verbose != 0) return;
    hipMemcpy(n_host, n, sizeof(int), hipMemcpyDeviceToHost);
    int true_n = min(*n_host, capacity);
    hipMemcpy(electrons_host, electrons, true_n * sizeof(Electron), hipMemcpyDeviceToHost);
    printf("Time %d, amount %d\n", t, *n_host);
    for(int i = 0; i < true_n; i++){
        printf("%d: (%.6f, %.6f) (%.6f, %.6f)\n", i, electrons_host[i].position.x, electrons_host[i].position.y, electrons_host[i].velocity.x, electrons_host[i].velocity.y);
    }
    image(true_n, electrons_host, t); // visualize a snapshot of the current positions of the particles     
    printf("\n");
}

void multiplyRun(int init_n, int capacity, int max_t, int mode, int verbose, int block_size) {
    printf("Multiply with\ninit n: %d\ncapacity: %d\nmax t: %d\nblock size: %d\n", init_n, capacity, max_t, block_size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    Electron* electrons_host = (Electron *)calloc(capacity, sizeof(Electron));
    for(int i=0; i<init_n; i++) {
        electrons_host[i].position = make_float3(250, 250, 1.0);
        electrons_host[i].weight = 1.0;
        electrons_host[i].timestamp = -1;
    }

    float delta_time = 0.1;

    Electron* electrons;
    hipMalloc(&electrons, capacity * sizeof(Electron));

    hipMemcpy(electrons, electrons_host, init_n * sizeof(Electron), hipMemcpyHostToDevice);

    int* n_host = (int*)malloc(sizeof(int));
    int* n;
    hipMalloc(&n, sizeof(int));
    *n_host = init_n;
    hipMemcpy(n, n_host, sizeof(int), hipMemcpyHostToDevice);

    if (verbose) printf("Time %d, amount %d\n", 0, *n_host);

    switch(mode){
        case 0: { // Normal
            printf("Multiply normal\n");
            hipEventRecord(start);
            for (int t = 1; t < max_t; t++){
                int num_blocks = (*n_host + block_size - 1) / block_size;
                updateNormal<<<num_blocks, block_size>>>(electrons, delta_time, n, min(*n_host, capacity), capacity);
                hipMemcpy(n_host, n, sizeof(int), hipMemcpyDeviceToHost);

                log(verbose, t, electrons_host, electrons, n_host, n, capacity);
            }
            hipEventRecord(stop);
            break;
        }
        case 1: { // Huge
            printf("Multiply huge\n");
            int num_blocks = (capacity + block_size - 1) / block_size;
            hipEventRecord(start);
            for (int t = 1; t < max_t; t++) {
                updateHuge<<<num_blocks, block_size>>>(electrons, delta_time, n, capacity, t);
                log(verbose, t, electrons_host, electrons, n_host, n, capacity);
            }
            hipEventRecord(stop);
            break;
        }
        case 2: { // Static simple
            printf("Multiply static simple\n");
            int num_blocks;
            hipDeviceGetAttribute(&num_blocks, hipDeviceAttributeMultiprocessorCount, 0);
            printf("Number of blocks: %d \n",num_blocks);
            hipEventRecord(start);
            for (int t = 1; t < max_t; t++) {
                updateStatic<<<num_blocks, block_size>>>(electrons, delta_time, n, capacity, t);
                log(verbose, t, electrons_host, electrons, n_host, n, capacity);
            }
            hipEventRecord(stop);
            break;
        }
        case 3: { // Static advanced
        
            printf("Multiply static advanced\n");
            int numBlocksPerSm = 0;
            // Number of threads my_kernel will be launched with
            int numThreads = block_size;
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, 0);  // What number should this actually be?
            hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, updateStatic, numThreads, 0);
            // launch
            dim3 dimBlock(numThreads, 1, 1);
            dim3 dimGrid(deviceProp.multiProcessorCount*numBlocksPerSm, 1, 1);
            printf("numBlocksPerSm: %d \n",numBlocksPerSm);
            printf("multiProcessorCount: %d \n",deviceProp.multiProcessorCount);

            hipEventRecord(start);
            for (int t = 1; t < max_t; t++) {
                void *kernelArgs[] = { &electrons, &delta_time, &n, &capacity, &t };
                hipLaunchCooperativeKernel((void*)updateStatic, dimGrid, dimBlock, kernelArgs);
                log(verbose, t, electrons_host, electrons, n_host, n, capacity);
            }
            hipEventRecord(stop);
            break;
        }
        case 4: { // Dynamic
            printf("Multiply dynamic not implemented\n");
            break;
        }
        default:
            break;
    }

    hipMemcpy(n_host, n, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(electrons_host, electrons, min(*n_host, capacity) * sizeof(Electron), hipMemcpyDeviceToHost);   
    hipEventSynchronize(stop); //skal det være her?

    float runtime_ms = 0;
    hipEventElapsedTime(&runtime_ms, start, stop);
    printf("Final amount of particles: %d\n", min(*n_host, capacity));
    printf("GPU time of program: %f ms\n", runtime_ms);
}
