#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cstring>
#include <math.h>
#include <stdexcept>
#include "mvp.h"

// __device__ hiprandState *d_rand_state;

__shared__ int i_block;
__shared__ int capacity;

__shared__ int n_block;
__shared__ int new_i_block;

__device__ void newRandState(hiprandState* d_rand_states, int i, int seed){
    // printf("New rand state for %d seed %d\n", i, seed);
    hiprand_init(1234, seed, 0, &d_rand_states[i]);
}

__device__ float randFloat(hiprandState* state, float min, float max){
    float rand = hiprand_uniform(state);
    rand *= (max - min + 0.999999);
    rand += min;
    return rand;
}

__device__ int randInt(hiprandState* state, int min, int max){
    float rand_float = randFloat(state, min, max);
    return (int)truncf(rand_float);
}

// Kernel for random numbers
__global__ void setup(Electron* d_electrons, hiprandState* d_rand_states, int init_n) {
    int i = threadIdx.x+blockDim.x*blockIdx.x;
    if (i >= init_n) return;
    newRandState(d_rand_states, i, i);
    d_electrons[i].position = make_float3(randFloat(&d_rand_states[i], 1, 499), randFloat(&d_rand_states[i], 1, 499), 1.0);
    d_electrons[i].weight = 1.0;
    d_electrons[i].timestamp = -1;
}

__device__ static int updateParticle(Electron* electron, Electron* new_electrons, float deltaTime, int* n, int capacity, float split_chance, hiprandState* d_rand_states, int i, int t){
    electron->velocity.y -= 9.82 * deltaTime * electron->weight;
    electron->position.y += electron->velocity.y * deltaTime;

    int new_i = -1;
    float rand = randFloat(&d_rand_states[i], 0, 100);
    // printf("%d: %.02f\n", i, rand);
    if (rand < split_chance) {
        if (*n < capacity) {
            // printf("n %d\n", *n);
            new_i = atomicAdd(n, 1);
        
            if (new_i < capacity){

                if (electron->velocity.x >= 0){
                    electron->velocity.x += 10;
                }
                else{
                    electron->velocity.x -= 10;
                }

                
                new_electrons[new_i].position.y = electron->position.y;
                new_electrons[new_i].velocity.y = electron->velocity.y;
                if (electron->velocity.x >= 0){
                    new_electrons[new_i].velocity.x = electron->velocity.x - 20;
                }
                else{
                    new_electrons[new_i].velocity.x = electron->velocity.x + 20;
                }
                new_electrons[new_i].position.x = electron->position.x + new_electrons[new_i].velocity.x * deltaTime;
                new_electrons[new_i].position.z = 1.0;
                new_electrons[new_i].velocity.z = 1.0;
                new_electrons[new_i].weight = electron->weight;
                new_electrons[new_i].creator = i;
                __threadfence();
                new_electrons[new_i].timestamp = t;
            }
        }
    }

    if (electron->position.y <= 0){
        electron->position.y = -electron->position.y;
        electron->velocity.y = -electron->velocity.y;
    }
    else if (electron->position.y >= 500){
        electron->position.y = 500 - (electron->position.y - 500);
        electron->velocity.y = -electron->velocity.y;
    }

    electron->position.x += electron->velocity.x * deltaTime;

    if (electron->position.x <= 0){
        electron->position.x = -electron->position.x;
        electron->velocity.x = -electron->velocity.x;
        electron->weight *= -1;
    }
    else if (electron->position.x >= 500){
        electron->position.x = 500 - (electron->position.x - 500);
        electron->velocity.x = -electron->velocity.x;
        electron->weight *= -1;
    }
    return new_i;
}

__device__ static void simulateNaive(Electron* d_electrons, Electron* new_electrons, float deltaTime, int* n, int capacity, float split_chance, hiprandState* d_rand_states, int i, int t){
    updateParticle(&d_electrons[i], new_electrons, deltaTime, n, capacity, split_chance, d_rand_states, i, t);
}

__device__ static void simulateMany(Electron* d_electrons, float deltaTime, int* n, int capacity, float split_chance, hiprandState* d_rand_states, int i, int start_t, int max_t){
    Electron electron = d_electrons[i];

    for(int t = start_t; t <= max_t; t++){
        int new_i = updateParticle(&electron, d_electrons, deltaTime, n, capacity, split_chance, d_rand_states, i, t);
        if(new_i != -1) {
            printf("Particle %d spawns particle %d\n", i, new_i);
            newRandState(d_rand_states, new_i, randInt(&d_rand_states[i], 0, 10000));
        }
    }
    d_electrons[i] = electron;
}

__global__ static void naive(Electron* d_electrons, float deltaTime, int* n, int start_n, int capacity, float split_chance, hiprandState* d_rand_states, int t) {
    //__shared__ char sharedMemory[sizeof(Electron) * 1024]; // Allocate raw shared memory
    //Electron* new_particles_block = reinterpret_cast<Electron*>(sharedMemory);
    __shared__ Electron new_particles_block[1024];

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadIdx.x == 0) n_block = 0;

    __syncthreads(); // Ensure construction is finished
    
    // The thread index has passed the number of d_electrons. Thread returns if all electron are being handled
    if (i >= start_n) return;

    simulateNaive(d_electrons, new_particles_block, deltaTime, &n_block, capacity, split_chance, d_rand_states, i, t);

    __syncthreads();

    if (threadIdx.x == 0){
        if (*n < capacity) new_i_block = atomicAdd(n, n_block);  // Avoid risk of n overflowing int max value
        else new_i_block = capacity;
    }

    __syncthreads();

    if (threadIdx.x >= n_block) return;
    int global_i = new_i_block + threadIdx.x;
    if (global_i >= capacity) return;
    newRandState(d_rand_states, global_i, randInt(&d_rand_states[new_particles_block[threadIdx.x].creator], 0, 10000));
    d_electrons[global_i] = new_particles_block[threadIdx.x];
}

__global__ static void cpuSynch(Electron* d_electrons, float deltaTime, int* n, int start_n, int offset, int capacity, float split_chance, hiprandState* d_rand_states, int max_t) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + offset;

    // The thread index has passed the number of d_electrons. Thread returns if all electron are being handled
    if (i >= start_n) return;
    simulateMany(d_electrons, deltaTime, n, capacity, split_chance, d_rand_states, i, max(1, d_electrons[i].timestamp + 1), max_t);
}

__global__ static void staticGpu(Electron* d_electrons, float deltaTime, int* n, int capacity, float split_chance, hiprandState* d_rand_states, int max_t, int sleep_time_ns, int* n_done) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int num_blocks = gridDim.x;
    int block_size = blockDim.x;
    int n_done_local = 0;

    for (int i = thread_id; i < capacity; i += num_blocks * block_size) {

        while(d_electrons[i].timestamp == 0) {
            if (n_done_local != 0){
                atomicAdd(n_done, n_done_local);
                n_done_local = 0;
            }
            int cur_n_done = *n_done;
            __threadfence();
            int cur_n = *n;
            if (cur_n==cur_n_done) return;
            __nanosleep(sleep_time_ns);
        }

        simulateMany(d_electrons, deltaTime, n, capacity, split_chance, d_rand_states, i, max(1, d_electrons[i].timestamp + 1), max_t);
        n_done_local++;
    }

    if (n_done_local != 0){
        atomicAdd(n_done, n_done_local);
    }

}

__global__ static void dynamicGpu(Electron* d_electrons, float deltaTime, int* n, int capacity, float split_chance, hiprandState* d_rand_states, int max_t, int sleep_time_ns, int* n_done, int* i_global) {

    while (true) {
        __syncthreads(); //sync threads seems to be able to handle threads being terminated
        if (threadIdx.x==0) {
            i_block = atomicAdd(i_global, blockDim.x);
        }
        __syncthreads();

        int i = i_block + threadIdx.x;

        if (i >= capacity) break;

        while (d_electrons[i].timestamp == 0) {
            int cur_n_done = *n_done;
            __threadfence();
            int cur_n = *n;
            if (cur_n==cur_n_done) return;
            __nanosleep(sleep_time_ns);
        }

        simulateMany(d_electrons, deltaTime, n, capacity, split_chance, d_rand_states, i, max(1, d_electrons[i].timestamp + 1), max_t);
        atomicAdd(n_done,1);

    }
}

static void log(int verbose, int t, Electron* electrons_host, Electron* electrons, int* n_host, int* n, int capacity){
    if (verbose == 0 || t % verbose != 0) return;
    hipMemcpy(n_host, n, sizeof(int), hipMemcpyDeviceToHost);
    int true_n = min(*n_host, capacity);
    hipMemcpy(electrons_host, electrons, true_n * sizeof(Electron), hipMemcpyDeviceToHost);
    printf("Time %d, amount %d\n", t, *n_host);
    for(int i = 0; i < true_n; i++){
        electrons_host[i].print(i);
    }
    image(true_n, electrons_host, t); // visualize a snapshot of the current positions of the particles     
    printf("\n");
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s \n", hipGetErrorString(error));
        throw runtime_error(hipGetErrorString(error));
        // Handle error appropriately
    }
}

RunData runMVP (int init_n, int capacity, int max_t, int mode, int verbose, int block_size, int sleep_time_ns, float delta_time, float split_chance) {
    printf("MVP with\ninit n: %d\ncapacity: %d\nmax t: %d\nblock size: %d\nsleep time: %d\ndelta time: %f\n", init_n, capacity, max_t, block_size, sleep_time_ns, delta_time);

    TimingData timing_data;
    timing_data.init_n = init_n;
    timing_data.iterations = max_t;
    timing_data.block_size = block_size;
    timing_data.sleep_time = sleep_time_ns;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    Electron* h_electrons = (Electron *)calloc(capacity, sizeof(Electron));
    Electron* d_electrons;
    hipMalloc(&d_electrons, capacity * sizeof(Electron));
    hipMemset(d_electrons, 0, capacity * sizeof(Electron));

    hiprandState* d_rand_states;
    hipMalloc(&d_rand_states, capacity * sizeof(hiprandState));
    setup<<<(init_n + block_size - 1) / block_size, block_size>>>(d_electrons, d_rand_states, init_n);

    int* n_host = (int*)malloc(sizeof(int));
    int* n;
    hipMalloc(&n, sizeof(int));
    *n_host = init_n;
    hipMemcpy(n, n_host, sizeof(int), hipMemcpyHostToDevice);

    int* n_done;
    hipMalloc(&n_done, sizeof(int));
    hipMemset(n_done, 0, sizeof(int));
    
    int* i_global;
    hipMalloc(&i_global, sizeof(int));
    hipMemset(i_global, 0, sizeof(int));

    switch(mode){
        case 0: { //Naive      
            timing_data.function = "Naive";
            hipEventRecord(start);
            for (int t = 1; t <= max_t; t++){
                int num_blocks = (min(*n_host, capacity) + block_size - 1) / block_size;
                naive<<<num_blocks, block_size>>>(d_electrons, delta_time, n, min(*n_host, capacity), capacity, split_chance, d_rand_states, t);
                log(verbose, t, h_electrons, d_electrons, n_host, n, capacity);
                hipMemcpy(n_host, n, sizeof(int), hipMemcpyDeviceToHost);
            }
            hipEventRecord(stop);
            break;
        }
        case 1: { //CPU Sync
            timing_data.function = "CPU Sync";
            hipEventRecord(start);
            int last_n = 0;  // The amount of particles present in last run. All of these have been fully simulated.
            while(min(*n_host, capacity) != last_n){  // Stop once nothing new has happened.
                int num_blocks = (min(*n_host, capacity) - last_n + block_size - 1) / block_size;  // We do not need blocks for the old particles.
                cpuSynch<<<num_blocks, block_size>>>(d_electrons, delta_time, n, min(*n_host, capacity), last_n, capacity, split_chance, d_rand_states, max_t);
                last_n = min(*n_host, capacity);  // Update last_n to the amount just run. NOT to the amount after this run (we don't know that amount yet).
                hipMemcpy(n_host, n, sizeof(int), hipMemcpyDeviceToHost);  // Now update to the current amount of particles.
            }
            hipEventRecord(stop);
            break;
        }
        case 2: { //Static
            timing_data.function = "Static";
            int num_blocks;
            hipDeviceGetAttribute(&num_blocks, hipDeviceAttributeMultiprocessorCount, 0);
            hipEventRecord(start);
            staticGpu<<<num_blocks, block_size>>>(d_electrons, delta_time, n, capacity, split_chance, d_rand_states, max_t, sleep_time_ns, n_done);
            hipEventRecord(stop);
            
            break;
        }
        case 3: { //Dynamic
            timing_data.function = "Dynamic";
            int num_blocks;
            hipDeviceGetAttribute(&num_blocks, hipDeviceAttributeMultiprocessorCount, 0);
            hipEventRecord(start);
            dynamicGpu<<<num_blocks, block_size>>>(d_electrons, delta_time, n, capacity, split_chance, d_rand_states, max_t, sleep_time_ns, n_done, i_global);
            hipEventRecord(stop);
            break;
        }
        default:
            break;
    }
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s \n", hipGetErrorString(error));
        throw runtime_error(hipGetErrorString(error));
        // Handle error appropriately
    }

    log(verbose, max_t, h_electrons, d_electrons, n_host, n, capacity);



    hipMemcpy(n_host, n, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_electrons, d_electrons, min(*n_host, capacity) * sizeof(Electron), hipMemcpyDeviceToHost);   

    hipEventSynchronize(stop);
    float runtime_ms = 0;
    hipEventElapsedTime(&runtime_ms, start, stop);
    printf("Final amount of particles: %d\n", min(*n_host, capacity));
    printf("GPU time of program: %f ms\n", runtime_ms);
    timing_data.time = runtime_ms;

    RunData run_data;
    run_data.timing_data = timing_data;
    run_data.final_n = min(*n_host, capacity);
    run_data.electrons = new Electron[capacity];
    memcpy(run_data.electrons, h_electrons, capacity * sizeof(Electron));

    free(n_host);
    free(h_electrons);
    hipFree(d_electrons);
    hipFree(n);
    hipFree(n_done);
    hipFree(i_global);
    hipFree(d_rand_states);

    return run_data;
}