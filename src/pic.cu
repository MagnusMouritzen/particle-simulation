#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstring>
#include <math.h>
#include <stdexcept>

#include "pic.h"

__shared__ int i_block;
__shared__ int capacity;

__shared__ int n_block;
__shared__ int new_i_block;


#define getGridCell(x,y,z) (((Cell*)((((char*)d_grid.ptr) + z * (d_grid.pitch * grid_size.y)) + y * d_grid.pitch))[x])


__device__ static void simulateMany(Electron* d_electrons, float deltaTime, int* n, int capacity, float split_chance, float remove_chance, hiprandState* rand_state, int i, int start_t, int poisson_timestep){
    Electron electron = d_electrons[i];

    for(int t = start_t; t <= poisson_timestep; t++){
        int new_i = updateParticle(&electron, d_electrons, deltaTime, n, capacity, split_chance, remove_chance, rand_state, i, t);
        if(new_i != -1 && new_i < capacity) {  // If a new particle was spawned and there is room for it.
            __threadfence();
            d_electrons[new_i].timestamp = t;
            printf("%d: (%d) NEW %d {%f}\n", i, t, new_i, d_electrons[new_i].position.x);
        }
        else if (electron.timestamp == DEAD){  // If particle is to be removed.
            printf("%d: (%d) DEAD\n", i, t);
            break;
        }
    }
    if (electron.timestamp != DEAD) electron.timestamp = -1;

    d_electrons[i] = electron;
}

__global__ static void poisson(Electron* d_electrons, float deltaTime, int* n, int capacity, float split_chance, float remove_chance, hiprandState* d_rand_states, int poisson_timestep, int sleep_time_ns, int* n_done, int* i_global) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    while (true) {
        __syncthreads(); //sync threads seems to be able to handle threads being terminated
        if (threadIdx.x==0) {
            i_block = atomicAdd(i_global, blockDim.x);
        }
        __syncthreads();

        int i = i_block + threadIdx.x;

        if (i >= capacity) break;

        while (d_electrons[i].timestamp == 0 || i >= *n) {
            int cur_n_done = *n_done;
            __threadfence();
            int cur_n = *n;
            if (cur_n==cur_n_done) return;
            __nanosleep(sleep_time_ns);
        }

        simulateMany(d_electrons, deltaTime, n, capacity, split_chance, remove_chance, &d_rand_states[thread_id], i, max(1, d_electrons[i].timestamp + 1), poisson_timestep);
        atomicAdd(n_done,1);

    }
}

__global__ static void remove_dead_particles(Electron* d_electrons_old, Electron* d_electrons_new, int* n, int start_n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= start_n) return;

    if (threadIdx.x == 0) n_block = 0;
    __syncthreads();

    int i_local = -1;
    if (d_electrons_old[i].timestamp != DEAD){
        i_local = atomicAdd(&n_block, 1);
    }

    __syncthreads();
    if (threadIdx.x == 0){
        i_block = atomicAdd(n, n_block);
    }
    __syncthreads();

    if (i_local == -1) return;
    d_electrons_new[i_block + i_local] = d_electrons_old[i];
}

__global__ static void particlesToGrid(hipPitchedPtr d_grid, Electron* d_electrons, int* n, int3 grid_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= *n) return;


    Electron electron = d_electrons[i];

    int x = electron.position.x/cell_size;
    int y = electron.position.y/cell_size;
    int z = electron.position.z/cell_size;

    getGridCell(x,y,z).charge += electron_charge;

}

__global__ static void gridToParticles(hipPitchedPtr d_grid, Electron* d_electrons, int* n, int3 grid_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= *n) return;

    Electron electron = d_electrons[i];

    int x = electron.position.x/cell_size;
    int y = electron.position.y/cell_size;
    int z = electron.position.z/cell_size;


    electron.acceleration =  getGridCell(x,y,z).acceleration;

    d_electrons[i] = electron;

}
__global__ void resetGrid(hipPitchedPtr d_grid, int3 grid_size) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    
    getGridCell(x,y,z).charge = 0;
}
__global__ void updateGrid(hipPitchedPtr d_grid, double electric_force_constant, int3 grid_size) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    // INSERT ELEMENT TO 3D ARRAY
    char* gridPtr = (char*)d_grid.ptr;
    size_t pitch = d_grid.pitch; // the number of bytes in a row of the array
    size_t slicePitch = pitch * grid_size.y; // the number of bytes pr slice

    char* slice = gridPtr + z * slicePitch; // get slice 

    Cell* row = (Cell*)(slice + y * pitch); // get row in slice


    double xAcc = 0;
    if (x != 0) xAcc -= row[x-1].charge;
    if (x != grid_size.x-1) xAcc += row[x+1].charge;
    xAcc *= electric_force_constant;

    double yAcc = 0;
    if (y != 0) yAcc -= (row-pitch)[x].charge;
    if (y != grid_size.y-1) yAcc += (row+pitch)[x].charge;
    yAcc *= electric_force_constant;

    double zAcc = 0;
    if (z != 0) zAcc -= (row-slicePitch)[x].charge;
    if (z != grid_size.z-1) zAcc += (row+slicePitch)[x].charge;
    zAcc *= electric_force_constant;

    
    row[x].acceleration = make_float3((float)xAcc, (float)yAcc, (float)zAcc);
}


RunData runPIC (int init_n, int capacity, int poisson_steps, int poisson_timestep, int mode, int verbose, int block_size, int sleep_time_ns, float split_chance, float remove_chance) {
    printf("MVP with\ninit n: %d\ncapacity: %d\npoisson steps: %d\npoisson_timestep: %d\nblock size: %d\nsleep time: %d\nsplit chance: %f\nremove chance: %f\n", init_n, capacity, poisson_steps, poisson_timestep, block_size, sleep_time_ns, split_chance, remove_chance);

    TimingData timing_data;
    timing_data.init_n = init_n;
    timing_data.iterations = poisson_steps;
    timing_data.block_size = block_size;
    timing_data.sleep_time = sleep_time_ns;
    timing_data.split_chance = split_chance;
    

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int num_blocks_pers;
    hipDeviceGetAttribute(&num_blocks_pers, hipDeviceAttributeMultiprocessorCount, 0);

    hiprandState* d_rand_states;
    hipMalloc(&d_rand_states, num_blocks_pers * block_size * sizeof(hiprandState));
    setup_rand<<<num_blocks_pers, block_size>>>(d_rand_states);  // This has to be done before setup_particles
    
    Electron* h_electrons = (Electron *)calloc(capacity, sizeof(Electron));
    Electron* d_electrons;
    hipMalloc(&d_electrons, 2 * capacity * sizeof(Electron));
    hipMemset(d_electrons, 0, 2 * capacity * sizeof(Electron));
    setup_particles<<<(init_n + block_size - 1) / block_size, block_size>>>(d_electrons, d_rand_states, init_n);

    int* n_host = (int*)malloc(sizeof(int));
    int* n;
    hipMalloc(&n, sizeof(int));
    *n_host = init_n;
    hipMemcpy(n, n_host, sizeof(int), hipMemcpyHostToDevice);

    int* n_done;
    hipMalloc(&n_done, sizeof(int));
    
    int* i_global;
    hipMalloc(&i_global, sizeof(int));


    hipExtent extent = make_hipExtent(Grid_Size.x * sizeof(Cell), Grid_Size.y, Grid_Size.z);
    hipPitchedPtr d_grid;
    hipMalloc3D(&d_grid, extent);

    dim3 dim_block(8,8,8);
    dim3 dim_grid(Grid_Size.x/dim_block.x, Grid_Size.y/dim_block.y, Grid_Size.z/dim_block.z);

    switch(mode){
        case 0: { // GOOD
            timing_data.function = "GOOD";
            hipEventRecord(start);

            int source_index = 0;
            int destination_index = 0;
            int num_blocks_all = (min(*n_host, capacity) + block_size - 1) / block_size;
            for (int t = 0; t < poisson_steps; t++)
            {
                source_index = (t % 2) * capacity;  // Flips between 0 and capacity.
                destination_index = ((t + 1) % 2) * capacity;  // Opposite of above.

                log(verbose, t, h_electrons, &d_electrons[source_index], n_host, n, capacity);
                hipMemset(n_done, 0, sizeof(int));
                hipMemset(i_global, 0, sizeof(int));

                resetGrid<<<dim_grid, dim_block>>>(d_grid, Grid_Size);
                particlesToGrid<<<num_blocks_all, block_size>>>(d_grid, d_electrons, n, Grid_Size);
                updateGrid<<<dim_grid, dim_block>>>(d_grid, Electric_Force_Constant, Grid_Size);
                gridToParticles<<<num_blocks_all, block_size>>>(d_grid, d_electrons, n, Grid_Size);

                poisson<<<num_blocks_pers, block_size>>>(&d_electrons[source_index], 0.1, n, capacity, split_chance, remove_chance, d_rand_states, poisson_timestep, sleep_time_ns, n_done, i_global);
                hipMemcpy(n_host, n, sizeof(int), hipMemcpyDeviceToHost);
                hipMemset(n, 0, sizeof(int));
                num_blocks_all = (min(*n_host, capacity) + block_size - 1) / block_size;
                if (*n_host == 0){
                    printf("Hit 0\n");
                    break;
                }
                remove_dead_particles<<<num_blocks_all, block_size>>>(&d_electrons[source_index], &d_electrons[destination_index], n, min(*n_host, capacity));
            }
            log(verbose, poisson_steps, h_electrons, &d_electrons[destination_index], n_host, n, capacity);
            
            
            hipEventRecord(stop);
            break;
        }
        default:
            break;
    }
    checkCudaError();

    hipMemcpy(n_host, n, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_electrons, d_electrons, min(*n_host, capacity) * sizeof(Electron), hipMemcpyDeviceToHost);   

    hipEventSynchronize(stop);
    float runtime_ms = 0;
    hipEventElapsedTime(&runtime_ms, start, stop);
    printf("Final amount of particles: %d\n", min(*n_host, capacity));
    printf("GPU time of program: %f ms\n", runtime_ms);
    timing_data.time = runtime_ms;
    timing_data.final_n = min(*n_host, capacity);

    RunData run_data;
    run_data.timing_data = timing_data;
    run_data.final_n = min(*n_host, capacity);
    run_data.electrons = new Electron[capacity];
    memcpy(run_data.electrons, h_electrons, capacity * sizeof(Electron));

    free(n_host);
    free(h_electrons);
    hipFree(d_electrons);
    hipFree(n);
    hipFree(n_done);
    hipFree(i_global);
    hipFree(d_rand_states);

    return run_data;
}


// LOOP OVER 3D ARRAY
// char* gridPtr = d_grid.ptr;
// size_t pitch = d_grid.pitch; // the number of bytes in a row of the array
// size_t slicePitch = pitch * grid_size.y; // the number of bytes pr slice
// for (int z = 0; z < grid_size.z; ++z) {
//     char* slice = gridPtr + z * slicePitch;
//     for (int y = 0; y < grid_size.y; ++y) {
//         Cell* row = (Cell*)(slice + y * pitch);
//         for (int x = 0; x < grid_size.x; ++x) {
//             Cell element = row[x];
//         }
//     }
// }


// INSERT ELEMENT TO 3D ARRAY
// char* gridPtr = d_grid.ptr;
// size_t pitch = d_grid.pitch; // the number of bytes in a row of the array
// size_t slicePitch = pitch * grid_size.y; // the number of bytes pr slice

// char* slice = gridPtr + z * slicePitch; // get slice 

// Cell* row = (Cell*)(slice + y * pitch); // get row in slice

// row[x].charge = 0;