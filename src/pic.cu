#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstring>
#include <math.h>
#include <stdexcept>

#include "pic.h"

__shared__ int i_block;
__shared__ int capacity;

__shared__ int n_block;
__shared__ int new_i_block;


__device__ static void simulateMany(Electron* d_electrons, float deltaTime, int* n, int capacity, float split_chance, float remove_chance, hiprandState* rand_state, int i, int start_t, int poisson_timestep){
    Electron electron = d_electrons[i];

    for(int t = start_t; t <= poisson_timestep; t++){
        int new_i = updateParticle(&electron, d_electrons, deltaTime, n, capacity, split_chance, remove_chance, rand_state, i, t);
        if(new_i != -1 && new_i < capacity) {  // If a new particle was spawned and there is room for it.
            __threadfence();
            d_electrons[new_i].timestamp = t;
            printf("%d: (%d) NEW %d {%f}\n", i, t, new_i, d_electrons[new_i].position.x);
        }
        else if (electron.timestamp == DEAD){  // If particle is to be removed.
            printf("%d: (%d) DEAD\n", i, t);
            break;
        }
    }
    if (electron.timestamp != DEAD) electron.timestamp = -1;

    d_electrons[i] = electron;
}

__global__ static void poisson(Electron* d_electrons, Cell* d_grid, float deltaTime, int* n, int capacity, float split_chance, float remove_chance, hiprandState* d_rand_states, int poisson_timestep, int sleep_time_ns, int* n_done, int* i_global) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    while (true) {
        __syncthreads(); //sync threads seems to be able to handle threads being terminated
        if (threadIdx.x==0) {
            i_block = atomicAdd(i_global, blockDim.x);
        }
        __syncthreads();

        int i = i_block + threadIdx.x;

        if (i >= capacity) break;

        while (d_electrons[i].timestamp == 0 || i >= *n) {
            int cur_n_done = *n_done;
            __threadfence();
            int cur_n = *n;
            if (cur_n==cur_n_done) return;
            __nanosleep(sleep_time_ns);
        }

        simulateMany(d_electrons, deltaTime, n, capacity, split_chance, remove_chance, &d_rand_states[thread_id], i, max(1, d_electrons[i].timestamp + 1), poisson_timestep);
        atomicAdd(n_done,1);

    }
}

__global__ static void remove_dead_particles(Electron* d_electrons_old, Electron* d_electrons_new, int* n, int start_n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= start_n) return;

    if (threadIdx.x == 0) n_block = 0;
    __syncthreads();

    int i_local = -1;
    if (d_electrons_old[i].timestamp != DEAD){
        i_local = atomicAdd(&n_block, 1);
    }

    __syncthreads();
    if (threadIdx.x == 0){
        i_block = atomicAdd(n, n_block);
    }
    __syncthreads();

    if (i_local == -1) return;
    d_electrons_new[i_block + i_local] = d_electrons_old[i];
}

__global__ void resetGrid(Cell[512][512][512] d_grid){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    d_grid[x][y][z].charge = 0;
}

__global__ void updateGrid(Cell[512][512][512] d_grid){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    
    double xAcc = 0;
    if (x != 0) xAcc -= Cell[x-1][y][z].charge;
    if (x != 511) xAcc += Cell[x+1][y][z].charge;
    xAcc *= electric_force_constant;

    double yAcc = 0;
    if (y != 0) yAcc -= Cell[x][y-1][z].charge;
    if (y != 511) yAcc += Cell[x][y+1][z].charge;
    yAcc *= electric_force_constant;

    double zAcc = 0;
    if (z != 0) zAcc -= Cell[x-1][y][z-1].charge;
    if (z != 511) zAcc += Cell[x][y][z+1].charge;
    zAcc *= electric_force_constant;

    Cell[x][y][z].acceleration = make_double3(xAcc, yAcc, zAcc);
}


RunData runPIC (int init_n, int capacity, int poisson_steps, int poisson_timestep, int mode, int verbose, int block_size, int sleep_time_ns, float split_chance, float remove_chance) {
    printf("MVP with\ninit n: %d\ncapacity: %d\npoisson steps: %d\npoisson_timestep: %d\nblock size: %d\nsleep time: %d\nsplit chance: %f\nremove chance: %f\n", init_n, capacity, poisson_steps, poisson_timestep, block_size, sleep_time_ns, split_chance, remove_chance);

    TimingData timing_data;
    timing_data.init_n = init_n;
    timing_data.iterations = poisson_steps;
    timing_data.block_size = block_size;
    timing_data.sleep_time = sleep_time_ns;
    timing_data.split_chance = split_chance;
    

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int num_blocks_pers;
    hipDeviceGetAttribute(&num_blocks_pers, hipDeviceAttributeMultiprocessorCount, 0);

    hiprandState* d_rand_states;
    hipMalloc(&d_rand_states, num_blocks_pers * block_size * sizeof(hiprandState));
    setup_rand<<<num_blocks_pers, block_size>>>(d_rand_states);  // This has to be done before setup_particles
    
    Electron* h_electrons = (Electron *)calloc(capacity, sizeof(Electron));
    Electron* d_electrons;
    hipMalloc(&d_electrons, 2 * capacity * sizeof(Electron));
    hipMemset(d_electrons, 0, 2 * capacity * sizeof(Electron));
    setup_particles<<<(init_n + block_size - 1) / block_size, block_size>>>(d_electrons, d_rand_states, init_n);

    int* n_host = (int*)malloc(sizeof(int));
    int* n;
    hipMalloc(&n, sizeof(int));
    *n_host = init_n;
    hipMemcpy(n, n_host, sizeof(int), hipMemcpyHostToDevice);

    int* n_done;
    hipMalloc(&n_done, sizeof(int));
    
    int* i_global;
    hipMalloc(&i_global, sizeof(int));


    Cell* d_grid;
    hipMalloc(&d_grid, grid_size.x * grid_size.y * grid_size.z * sizeof(Cell));

    dim3 dim_block(8,8,8);
    dim3 dim_grid(grid_size.x/dim_block.x, grid_size.y/dim_block.y, grid_size.z/dim_block.z);

    switch(mode){
        case 0: { // GOOD
            timing_data.function = "GOOD";
            hipEventRecord(start);

            int source_index = 0;
            int destination_index = 0;
            int num_blocks_all = (min(*n_host, capacity) + block_size - 1) / block_size;
            for (int t = 0; t < poisson_steps; t++)
            {
                source_index = (t % 2) * capacity;  // Flips between 0 and capacity.
                destination_index = ((t + 1) % 2) * capacity;  // Opposite of above.

                log(verbose, t, h_electrons, &d_electrons[source_index], n_host, n, capacity);
                hipMemset(n_done, 0, sizeof(int));
                hipMemset(i_global, 0, sizeof(int));

                resetGrid<<<dim_grid, dim_block>>>(d_grid);
                particlesToGrid<<<num_blocks_all, block_size>>>(d_grid, d_electrons);
                updateGrid<<<dim_grid, dim_block>>>(d_grid);
                gridToParticles<<<num_blocks_all, block_size>>>(d_grid, d_electrons);

                poisson<<<num_blocks_pers, block_size>>>(&d_electrons[source_index], d_grid, 0.1, n, capacity, split_chance, remove_chance, d_rand_states, poisson_timestep, sleep_time_ns, n_done, i_global);
                hipMemcpy(n_host, n, sizeof(int), hipMemcpyDeviceToHost);
                hipMemset(n, 0, sizeof(int));
                num_blocks_all = (min(*n_host, capacity) + block_size - 1) / block_size;
                if (*n_host == 0){
                    printf("Hit 0\n");
                    break;
                }
                remove_dead_particles<<<num_blocks_all, block_size>>>(&d_electrons[source_index], &d_electrons[destination_index], n, min(*n_host, capacity));
            }
            log(verbose, poisson_steps, h_electrons, &d_electrons[destination_index], n_host, n, capacity);
            
            
            hipEventRecord(stop);
            break;
        }
        default:
            break;
    }
    checkCudaError();

    hipMemcpy(n_host, n, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_electrons, d_electrons, min(*n_host, capacity) * sizeof(Electron), hipMemcpyDeviceToHost);   

    hipEventSynchronize(stop);
    float runtime_ms = 0;
    hipEventElapsedTime(&runtime_ms, start, stop);
    printf("Final amount of particles: %d\n", min(*n_host, capacity));
    printf("GPU time of program: %f ms\n", runtime_ms);
    timing_data.time = runtime_ms;
    timing_data.final_n = min(*n_host, capacity);

    RunData run_data;
    run_data.timing_data = timing_data;
    run_data.final_n = min(*n_host, capacity);
    run_data.electrons = new Electron[capacity];
    memcpy(run_data.electrons, h_electrons, capacity * sizeof(Electron));

    free(n_host);
    free(h_electrons);
    hipFree(d_electrons);
    hipFree(n);
    hipFree(n_done);
    hipFree(i_global);
    hipFree(d_rand_states);

    return run_data;
}