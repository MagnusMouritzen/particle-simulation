#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "utility.h"
#include "pic_simulation.h"

__device__ static void simulate(Electron* electrons, float deltaTime, int* n, int capacity, int i, int t){
    electrons[i].velocity.y -= 9.82 * deltaTime * electrons[i].weight;
    electrons[i].position.y += electrons[i].velocity.y * deltaTime;

    if (electrons[i].position.y <= 0){
        electrons[i].position.y = -electrons[i].position.y;
        electrons[i].velocity.y = -electrons[i].velocity.y;

        int new_i = atomicAdd(n, 1);
        if (new_i < capacity){
            if (electrons[i].velocity.x >= 0){
                electrons[i].velocity.x += 10;
            }
            else{
                electrons[i].velocity.x -= 10;
            }

            //printf("Particle %d spawns particle %d\n", i, new_i);
            electrons[new_i].position.y = electrons[i].position.y;
            electrons[new_i].velocity.y = electrons[i].velocity.y;
            if (electrons[i].velocity.x >= 0){
                electrons[new_i].velocity.x = electrons[i].velocity.x - 20;
            }
            else{
                electrons[new_i].velocity.x = electrons[i].velocity.x + 20;
            }
            electrons[new_i].position.x = electrons[i].position.x + electrons[new_i].velocity.x * deltaTime;
            electrons[new_i].timestamp = t;
            electrons[new_i].weight = electrons[i].weight;
        }
    }
    else if (electrons[i].position.y >= 500){
        electrons[i].position.y = 500 - (electrons[i].position.y - 500);
        electrons[i].velocity.y = -electrons[i].velocity.y;
    }

    electrons[i].position.x += electrons[i].velocity.x * deltaTime;

    if (electrons[i].position.x <= 0){
        electrons[i].position.x = -electrons[i].position.x;
        electrons[i].velocity.x = -electrons[i].velocity.x;
        electrons[i].weight *= -1;
    }
    else if (electrons[i].position.x >= 500){
        electrons[i].position.x = 500 - (electrons[i].position.x - 500);
        electrons[i].velocity.x = -electrons[i].velocity.x;
        electrons[i].weight *= -1;
    }
}

__global__ static void updateStatic(Electron* electrons, float deltaTime, int* n, int capacity, int t, int max_t) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int num_blocks = gridDim.x;
    int block_size = blockDim.x;

    for (int i = thread_id; i < min(*n, capacity); i += num_blocks * block_size) {
        // The thread index has passed the number of electrons. Thread returns if all electron are being handled
        if (electrons[i].timestamp == t || electrons[i].timestamp == 0) return;

        for (int j = 1 ; j < max_t ; j++) {
            simulate(electrons, deltaTime, n, capacity, i, t);
        }
    }
}

static void log(int verbose, int t, Electron* electrons_host, Electron* electrons, int* n_host, int* n, int capacity){
    if (verbose == 0 || t % verbose != 0) return;
    hipMemcpy(n_host, n, sizeof(int), hipMemcpyDeviceToHost);
    int true_n = min(*n_host, capacity);
    hipMemcpy(electrons_host, electrons, true_n * sizeof(Electron), hipMemcpyDeviceToHost);
    printf("Time %d, amount %d\n", t, *n_host);
    for(int i = 0; i < true_n; i++){
        printf("%d: (%.6f, %.6f) (%.6f, %.6f)\n", i, electrons_host[i].position.x, electrons_host[i].position.y, electrons_host[i].velocity.x, electrons_host[i].velocity.y);
    }
    image(true_n, electrons_host, t); // visualize a snapshot of the current positions of the particles     
    printf("\n");
}

void runPIC(int init_n, int capacity, int max_t, int verbose, int block_size) {
    printf("PIC with\ninit n: %d\ncapacity: %d\nmax t: %d\nblock size: %d\n", init_n, capacity, max_t, block_size);
    
    Electron* electrons_host = (Electron *)calloc(capacity, sizeof(Electron));
    for(int i=0; i<init_n; i++) {
        electrons_host[i].position = make_float3(250, 250, 1.0);
        electrons_host[i].weight = 1.0;
        electrons_host[i].timestamp = -1;
    }

    float delta_time = 0.1;

    Electron* electrons;
    hipMalloc(&electrons, capacity * sizeof(Electron));

    hipMemcpy(electrons, electrons_host, capacity * sizeof(Electron), hipMemcpyHostToDevice);

    int* n_host = (int*)malloc(sizeof(int));
    int* n;
    hipMalloc(&n, sizeof(int));
    *n_host = init_n;
    hipMemcpy(n, n_host, sizeof(int), hipMemcpyHostToDevice);
    
    if (verbose) printf("Time %d, amount %d\n", 0, *n_host);


    printf("Multiply static advanced\n");
    int numBlocksPerSm = 0;
    // Number of threads my_kernel will be launched with
    int numThreads = block_size;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);  // What number should this actually be?
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, updateStatic, numThreads, 0);
    // launch
    dim3 dimBlock(numThreads, 1, 1);
    dim3 dimGrid(deviceProp.multiProcessorCount*numBlocksPerSm, 1, 1);
    printf("numBlocksPerSm: %d \n",numBlocksPerSm);
    printf("multiProcessorCount: %d \n",deviceProp.multiProcessorCount);
    
    for (int i = 0; i<*n; i++) {
        void *kernelArgs[] = { &electrons, &delta_time, &n, &capacity, &max_t };
        hipLaunchCooperativeKernel((void*)updateStatic, dimGrid, dimBlock, kernelArgs);
        // log(verbose, t, electrons_host, electrons, n_host, n, capacity);
    }
    hipMemcpy(n_host, n, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(electrons_host, electrons, min(*n_host, capacity) * sizeof(Electron), hipMemcpyDeviceToHost);

    float runtime_ms = 0;
    printf("Final amount of particles: %d\n", min(*n_host, capacity));
    printf("GPU time of program: %f ms\n", runtime_ms);

}