#include "hip/hip_runtime.h"
#include "random.h" 
#include <stdio.h>

__device__ void newRandState(hiprandState* state, int seed){
    hiprand_init(39587 + seed, 0, 0, state);  // Keep base seed below 47483647
}

__device__ float randFloat(hiprandState* state, float min, float max){
    float rand = hiprand_uniform(state);
    rand *= (max - min);
    rand += min;
    return rand;
}

__device__ int randInt(hiprandState* state, int min, int max){
    float rand = hiprand_uniform(state);
    rand *= (max - min + 0.999999);
    rand += min;
    return (int)truncf(rand);
}
