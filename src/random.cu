#include "hip/hip_runtime.h"
#include "random.h" 

__device__ void newRandState(hiprandState* rand_state, int seed){
    hiprand_init(39587 + seed, 0, 0, rand_state);
}

__device__ float randFloat(hiprandState* state, float min, float max){
    float rand = hiprand_uniform(state);
    rand *= (max - min);
    rand += min;
    return rand;
}

__device__ int randInt(hiprandState* state, int min, int max){
    float rand = hiprand_uniform(state);
    rand *= (max - min + 0.999999);
    rand += min;
    return (int)truncf(rand);
}

__global__ void setup_rand(hiprandState* d_rand_states) {
    int i = threadIdx.x+blockDim.x*blockIdx.x;
    newRandState(&d_rand_states[i], i);
}