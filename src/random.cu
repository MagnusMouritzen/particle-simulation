#include "hip/hip_runtime.h"
#include "random.h" 

__device__ void newRandState(hiprandState* rand_state, int sequence){
    hiprand_init(39587, sequence, 0, rand_state);
}

__device__ float randFloat(hiprandState* state, float min, float max){
    float rand = hiprand_uniform(state);
    rand *= (max - min);
    rand += min;
    return rand;
}

__device__ int randInt(hiprandState* state, int min, int max){
    float rand = hiprand_uniform(state);
    rand *= (max - min + 0.999999);
    rand += min;
    return (int)truncf(rand);
}

__global__ void setup_rand(hiprandState* d_rand_states) {
    int i = threadIdx.x+blockDim.x*blockIdx.x;
    newRandState(&d_rand_states[i], i);
}
