#include "hip/hip_runtime.h"
#include "random.h" 

__device__ void newRandState(hiprandState* rand_state, int seed){
    hiprand_init(39587 + seed, 0, 0, rand_state);  // Keep base seed below 47483647
}

__device__ float randFloat(hiprandState* state, float min, float max){
    float rand = hiprand_uniform(state);
    rand *= (max - min);
    rand += min;
    return rand;
}

__device__ int randInt(hiprandState* state, int min, int max){
    float rand = hiprand_uniform(state);
    rand *= (max - min + 0.999999);
    rand += min;
    return (int)truncf(rand);
}
