#include "utility.h"

float randomFloat() {
    return (float)(rand()) / (float)(RAND_MAX);
}

int randomInt(int a, int b)
{
    if (a > b)
        return randomInt(b, a);
    if (a == b)
        return a;
    return a + (rand() % (b - a));
}

float randomFloat(int a, int b)
{
    if (a > b)
        return randomFloat(b, a);
    if (a == b)
        return a;
 
    return (float)randomInt(a, b) + randomFloat();
}

